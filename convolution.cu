#include <hipDNN.h>
#include <iostream>
#include <string>
#include <cstring>
#include <dirent.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <opencv2/opencv.hpp>

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }




cv::Mat load_image(const char* image_path) {
    cv::Mat image = cv::imread(image_path, CV_LOAD_IMAGE_COLOR);
    image.convertTo(image, CV_32FC3);
    cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
    return image;
}

void save_image(const char* output_filename,
        float* buffer,
        int height,
        int width) {
    cv::Mat output_image(height, width, CV_32FC3, buffer);
    // Make negative values zero.
    cv::threshold(output_image,
                    output_image,
                    /*threshold=*/0,
                    /*maxval=*/0,
                    cv::THRESH_TOZERO);
    
    cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
    output_image.convertTo(output_image, CV_8UC3);
    cv::imwrite(output_filename, output_image);
} 

void convolution(const char* img_path,const char* output_path){

    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));

    cv::Mat image = load_image(img_path);

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                      /*format=*/HIPDNN_TENSOR_NHWC,
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*batch_size=*/1,
                                      /*channels=*/3,
                                      /*image_height=*/image.rows,
                                      /*image_width=*/image.cols));

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NHWC,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*batch_size=*/1,
                                        /*channels=*/3,
                                        /*image_height=*/image.rows,
                                        /*image_width=*/image.cols));


    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*out_channels=*/3,
                                        /*in_channels=*/3,
                                        /*kernel_height=*/3,
                                        /*kernel_width=*/3));

    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                            /*pad_height=*/1,
                                            /*pad_width=*/1,
                                            /*vertical_stride=*/1,
                                            /*horizontal_stride=*/1,
                                            /*dilation_height=*/1,
                                            /*dilation_width=*/1,
                                            /*mode=*/HIPDNN_CROSS_CORRELATION,
                                            /*computeType=*/HIPDNN_DATA_FLOAT));

    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                                    input_descriptor,
                                                    kernel_descriptor,
                                                    convolution_descriptor,
                                                    output_descriptor,
                                                    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                    /*memoryLimitInBytes=*/0,
                                                    &convolution_algorithm));

    //Asking how much memory cudnn needs to do the convolution operation
    size_t workspace_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                    input_descriptor,
                                                    kernel_descriptor,
                                                    convolution_descriptor,
                                                    output_descriptor,
                                                    convolution_algorithm,
                                                    &workspace_bytes));
    std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB" << std::endl;

    void* d_workspace = nullptr;

    hipMalloc(&d_workspace,workspace_bytes);

    int batch_size,channels,height,width;
    
    hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,input_descriptor,kernel_descriptor,&batch_size,&channels,&height,&width);

    std::cout << "Output dimensions" << std::endl << std::endl; 
    std::cout << "Batch Size: " << batch_size << std::endl;
    std::cout << "Channels: " << channels << std::endl;
    std::cout << "Height: " << height << std::endl;
    std::cout << "Width: " << width << std::endl << std::endl;
    std::cout << "NCHW: " << batch_size << " x " << channels << " x " << height << " x " << width << std::endl;

    int image_bytes = batch_size * channels * height * width * sizeof(float);

    float *d_input = nullptr, *d_output = nullptr, *d_kernel = nullptr;

    //Copy input tensor to device memory
    hipMalloc(&d_input,image_bytes);
    hipMemcpy(d_input,image.ptr<float>(0),image_bytes,hipMemcpyHostToDevice);

    //Allocate memory for the output tensor in the device and initialize the bytes with 0 value
    hipMalloc(&d_output, image_bytes);
    hipMemset(d_output,0,image_bytes);

    //The convolution kernel
    const float kernel_template[3][3] = {
        {1,  1, 1},
        {1, -8, 1},
        {1,  1, 1}
    };

    float h_kernel[3][3][3][3];

    //Apply kernel template for all 3 kernels and 3 channels(RGB)
    for(int kernel = 0; kernel < 3; kernel++){
        for(int channel = 0; channel < 3; channel++){
            for(int row = 0; row < 3; row++){
                for(int column = 0; column < 3; column++){
                    h_kernel[kernel][channel][row][column] = kernel_template[row][column];
                }
            }
        }
    }

    //Allocate memory to the kernel in the device
    hipMalloc(&d_kernel,sizeof(h_kernel));
    hipMemcpy(d_kernel,h_kernel,sizeof(h_kernel),hipMemcpyHostToDevice);

    //Convolution (finally!)
    const float alpha = 1, beta = 0;
    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                   &alpha,
                                   input_descriptor,
                                   d_input,
                                   kernel_descriptor,
                                   d_kernel,
                                   convolution_descriptor,
                                   convolution_algorithm,
                                   d_workspace,
                                   workspace_bytes,
                                   &beta,
                                   output_descriptor,
                                   d_output));
    
    
                                   float* h_output = new float[image_bytes];
    
    //Copy output to host and release device memory
    hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);
    
    save_image(output_path, h_output, height, width);

    //free(output_file);
    //free(output_dir);
    //free(output_path);

    delete[] h_output;

    hipFree(d_kernel);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

    hipdnnDestroy(cudnn);


}

int main(int argc, char** argv){
 
    DIR *dir;
    struct dirent *ent;

    char* dirname = "./inputs";
    std::string input_path,output_path;

    if ((dir = opendir (dirname)) != NULL) {
        
        while ((ent = readdir (dir)) != NULL) {

            std::string strdir = dirname;
            std::string strfile = ent->d_name;

            if(strfile.find(".png") != std::string::npos){                
                //path = (char*)malloc((strlen(dirname)+strlen(ent->d_name) + 1)*sizeof(char));

                //strcpy(path,dirname);
                //strcpy(path,"/");
                //strcpy(path,ent->d_name);

                input_path = strdir + "/" + strfile;
                output_path = "./outputs/" + strfile;

                std::cout << strfile << std::endl << std::endl;

                convolution(input_path.c_str(),output_path.c_str());

            }

            
        }
        
        closedir (dir);
        
    } else {
        
        /* could not open directory */
        perror ("");
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;

    //convolution("./inputs/nvidia.png","./outputs/cudnn-1.png");
    

}
